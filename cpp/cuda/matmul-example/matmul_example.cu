#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "matmul_example.h"

void cuda_check(hipError_t status, const char* action = NULL,
                const char* file = NULL, int32_t line = 0) {
  // check for cuda errors

  if (status != hipSuccess) {
    printf("CUDA error occurred: %s\n", hipGetErrorString(status));
    if (action != NULL)
      printf("While running %s   (file %s, line %d)\n", action, file, line);
    exit(1);
  }
}

#define CUDA_CHECK(action) cuda_check(action, #action, __FILE__, __LINE__)

__global__ void matrix_mul_kernel(const float* A, const float* B, float* C,
                                  int size) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = col / size;
  col %= size;

  if (row < size && col < size) {
    float sum = 0.0f;
    for (int k = 0; k < size; ++k) {
      sum += A[row * size + k] * B[k * size + col];
    }
    C[row * size + col] = sum;
  }
}

void matmul(float* A, float* B, float* C, int size) {
  int m = size * size;
  float *d_A, *d_B, *d_C;
  CUDA_CHECK(hipMalloc(&d_A, m * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_B, m * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_C, m * sizeof(float)));

  // Copy data from host to device
  CUDA_CHECK(hipMemcpy(d_A, A, m * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_B, B, m * sizeof(float), hipMemcpyHostToDevice));

  // Define grid and block dimensions
  int blockSize = 256;
  int numBlocks = (m + blockSize - 1) / blockSize;
  // Launch the kernel
  matrix_mul_kernel<<<numBlocks, blockSize>>>(d_A, d_B, d_C, size);

  CUDA_CHECK(hipMemcpy(C, d_C, m * sizeof(float), hipMemcpyDeviceToHost));

  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_B));
  CUDA_CHECK(hipFree(d_C));
}
